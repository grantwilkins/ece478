
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <assert.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#define THREADS_PER_BLOCK 1024

//Kernel function, create a
__global__ void GPU_big_dot(float *A, float *B, float *C, long long N) {
	
	__shared__ float device_data[THREADS_PER_BLOCK];
	unsigned int idx, i;
	float mult;
	idx = threadIdx.x + blockIdx.x * blockDim.x; // Normal indexing

	device_data[threadIdx.x] = A[idx] * B[idx];

	__syncthreads();

	//Reduction done by the root thread in block
	if(threadIdx.x == 0)
	{
		mult = 0.0;
		for(i = 0; i < THREADS_PER_BLOCK; i++)
			mult += device_data[i];
		atomicAdd(C, mult);
	}
}	



// Code provided by Dr. Jin
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}

// Code provided by Dr. Jin
long long stop_timer(long long start_time, char *name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
	printf("%s: %.5f sec\n", name, ((float) (end_time-start_time)) / (1000 * 1000));
	return (end_time-start_time);
}

// Code provided by Dr. Jin
float * get_random_vector(int N) {
      if (N < 1) exit(1);
      // Allocate memory for the vector
      float *V = (float *) malloc(N * sizeof(float));
      if (V == NULL) exit(2);
      // Populate the vector with random numbers
      for (int i = 0; i < N; i++) V[i] = (float) rand() / (float) rand();
      // Return the randomized vector
      return V;
}

float CPU_big_dot(float *A, float *B, long long N) {

	int i;
	float c = 0.0;
	for(i = 0; i < N; i++)
	{
		c += A[i] * B[i];
	}
	return c;
}



int main(int argc, char ** argv) {

	long long N = 100*512*512;
	if(argc == 2)
	{
		N = atoi(argv[1]); // Allow user to set number of elements
	}
	else
	{
		printf("ATTN: Usage: ./dot-product <NUM_ELEMS>\n");
		printf("Proceeding with default N = %lld\n\n", N);
	}

	//Initial variables
	long long start_cpu = 0, start_gpu = 0, stop_cpu = 0, stop_gpu1 = 0, stop_gpu2 = 0, stop_gpu3 = 0;
	char name_cpu[] = "CPU: Tcpu", name_gpu[] = "GPU Kernel: Tker", name_gpu2[] = "GPU Memcpy: Tmem";
	float *v1, *v2, *result_gpu, result_cpu = 0.0; // host copies
	float *device_v1, *device_v2, *device_result_gpu; // device copies
	int size = N * sizeof(float);

	//Allocate memory and generate random vectors;
	result_gpu = (float *) malloc(sizeof(float));
	*result_gpu = 0.0;
	v1 = get_random_vector(N);
	v2 = get_random_vector(N);

	//Allocate device memory
	hipMalloc((void **)&device_v1, size);
	hipMalloc((void **)&device_v2, size);
	hipMalloc((void **)&device_result_gpu, sizeof(float));
	hipMemset(device_result_gpu, 0.0, sizeof(float));

	//Compute CPU
	start_cpu = start_timer();
	result_cpu = CPU_big_dot(v1, v2, N);
	stop_cpu = stop_timer(start_cpu, name_cpu);


	//This is to ensure that with some N < THREADS_PER_BLOCK we will
	//still have a non-zero number of blocks.
    	dim3 numBlocks((N+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK);
	
	start_gpu = start_timer();
	//Copy the inputs to device
	hipMemcpy(device_v1, v1, size, hipMemcpyHostToDevice);
	hipMemcpy(device_v2, v2, size, hipMemcpyHostToDevice);
    	stop_gpu1 = stop_timer(start_gpu, name_gpu2);

    	//Compute GPU
    	start_gpu = start_timer();
	GPU_big_dot<<<numBlocks,THREADS_PER_BLOCK>>>(device_v1, device_v2, device_result_gpu, N);
    	stop_gpu2 = stop_timer(start_gpu, name_gpu);

    	//Send data back
    	start_gpu = start_timer();
	hipMemcpy(result_gpu,device_result_gpu,sizeof(float),hipMemcpyDeviceToHost);
	stop_gpu3 = stop_timer(start_gpu, name_gpu2);
	

	//STATS REGION
	printf("\nCPU Result: %e\n", result_cpu);
	printf("GPU Result: %e\n", *result_gpu);
	printf("\nSpeedup (with data transfer): %lf\n", (float)(stop_cpu)/ ((float)(stop_gpu1 + stop_gpu2 + stop_gpu3)));
    	printf("Speedup (just kernel): %lf\n", (float) stop_cpu / (float) stop_gpu2);
	printf("\nRelative Error: %lf%%\n", 1e2*abs(*result_gpu - result_cpu) / (result_cpu));
	//Cleanup and exit
	hipFree(device_v1);
	hipFree(device_v2);
	hipFree(device_result_gpu);
	free(v1);
	free(v2);
	free(result_gpu);
	return 0;
}
